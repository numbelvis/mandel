#include "hip/hip_runtime.h"


/*

	Kernel for use with ManagedCuda

*/
__global__ void kernel(int *c, int xsize, double y_scale, double y_base, double x_scale, double x_base, int max)
{
	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = xsize * idx_y + idx_x;

	double x0 = (x_base + ((double)idx_x * x_scale));
	double y0 = (y_base + ((double)idx_y * y_scale));
	double x = 0;
	double y = 0;
	int ii = 0;
	double temp = 0;
	while (ii < max && x * x + y * y < 4)
	{
		temp = x * x - y * y + x0;
		y = x * 2 * y + y0;
		x = temp;
		ii++;
	}


	c[idx] = ii;
}

int main()
{
	return 0;
}