#include <stdio.h>
#include <inttypes.h>
#include "hip/hip_runtime.h"



#ifndef CONFIG_DEFINED
#define CONFIG_DEFINED

// Number of real digits per Digit stored.
static const int PRECISION = 4;


// 10 ^ PRECISION: Used in many calcs.
static const int MAGNITUDE = 10000;


// Maximum number of Digits that will be stored per integer or decimal.
static const int DIGITS = 25;
static const int DMO = DIGITS - 1;			// DIGITS MINUS ONE.  Used in loops and indexing.

static const int HUGE_DIGITS = DIGITS * 2; // DIGITS TIMES TWO
static const int HDMO = HUGE_DIGITS - 1;			// HUGE DIGITS MINUS ONE.  Used in loops and indexing.

#endif



#ifndef TYPES_DEFINED
#define TYPES_DEFINED

// An arbitrary precision integer.
struct ArbInt
{
	bool sign;
	uint16_t digits[DIGITS];
};

// An arbitrary int used for multiplication calcs.
struct ArbHugeInt
{
	bool sign;
	uint16_t digits[DIGITS * 2];
};

// An arbitrary precision decimal.
struct ArbDec
{
	bool sign = true;
	uint16_t digits[DIGITS];
	int decpos = 0;
};

#endif

__device__ void PrintArb(ArbDec *num)
{
	if (num->sign == false)
		printf("-");

	bool started = false;
	bool first = true;
	for (int ii = 0; ii < num->decpos; ii++)
	{
		if (started == false && num->digits[ii] != 0)
			started = true;

		if (started)
		{
			if (first)
			{
				first = false;
				printf("%d", num->digits[ii]);
			}
			else
				printf("%04d", num->digits[ii]);
		}
	}

	int endat = num->decpos;
	for (int ii = DMO; ii >= num->decpos; ii--)
	{
		if (num->digits[ii] != 0)
		{
			endat = ii;
			if (endat < 0)
				endat = 0;

			break;
		}
	}

	if (endat > num->decpos - 1)
	{
		printf(".");
		for (int ii = num->decpos; ii <= endat; ii++)
		{
			printf("%04d", num->digits[ii]);
		}
	}
}


/**************** Zero-related ****************************************************/

/*
Zero out all elements in an Arb number.
*/
__device__ void ZeroOut(ArbInt *arb)
{
	for (int kk = 0; kk < DIGITS; kk++)
		arb->digits[kk] = 0;
}

__device__ void ZeroOut(ArbDec *arb)
{
	for (int kk = 0; kk < DIGITS; kk++)
		arb->digits[kk] = 0;
}

__device__ void ZeroOut(ArbHugeInt *arb)
{
	for (int kk = 0; kk < HUGE_DIGITS; kk++)
		arb->digits[kk] = 0;
}


/*
Retrieve a zero value.
*/
__device__ ArbInt GetZeroInt()
{
	struct ArbInt arb;
	arb.sign = true;
	ZeroOut(&arb);
	return arb;
}

__device__ ArbDec GetZeroDec()
{
	struct ArbDec arb;
	arb.sign = true;
	arb.decpos = 0;
	ZeroOut(&arb);
	return arb;
}



/******************* Convert from regular number types or between arb types ******************************/

__device__ ArbInt ArbFromInteger(int num)
{
	struct ArbInt arb = GetZeroInt();

	// Determine the sign.
	arb.sign = num >= 0;

	// Get rid of the sign.
	if (num < 0)
		num = -num;

	for (int ii = DIGITS - 1; ii >= 0; ii--)
	{
		int this_digit = num % MAGNITUDE;
		arb.digits[ii] = this_digit;
		num = num / MAGNITUDE;
	}

	return arb;
}



__device__ ArbDec ArbFromDouble(double num)
{
	// Start with an int.
	int as_int = (int)num;
	ArbInt i = ArbFromInteger(as_int);

	int zero = -1;
	double rest = num - (double)as_int;

	struct ArbDec arb = GetZeroDec();

	arb.sign = num >= 0;

	for (int kk = 0; kk < DIGITS; kk++)
	{
		arb.digits[kk] = 0;
		if (i.digits[kk] != 0)
			zero = kk;
	}

	if (zero == -1)
	{
		// This number has no whole part.
		arb.decpos = 0;
		zero = DIGITS;
	}

	// Copy the int digits over
	int numofnonzero = DIGITS - zero;
	for (int kk = 0; kk < numofnonzero; kk++)
	{
		arb.digits[kk] = i.digits[kk + zero];
	}

	arb.decpos = numofnonzero;


	// We go 8 * magnitude deep.

	// Get rid of the sign
	if (rest < 0)
		rest = -rest;

	for (int ii = 0; ii < 8; ii++)
	{
		rest = rest * MAGNITUDE;
		int part = (int)rest;
		arb.digits[ii + numofnonzero] = part;

		rest = rest - (double)part;
	}

	return arb;
}


__device__ ArbDec Construct(bool sign, int decpos, uint16_t *copyFromDigits)
{
	struct ArbDec arb;
	arb.sign = sign;
	arb.decpos = decpos;

	for (int kk = 0; kk < DIGITS; kk++)
		arb.digits[kk] = copyFromDigits[kk];

	return arb;
}



/************************ Comparison of Arb Number ******************************/

__device__ bool GreaterThan(ArbDec *a, ArbDec *b)
{
	if (a->decpos == b->decpos)
	{

		for (int ii = 0; ii < DIGITS; ii++)
			if (a->digits[ii] != b->digits[ii])
				return a->digits[ii] > b->digits[ii];
	}
	else
	{
		return a->decpos > b->decpos;
	}

	return false;
}


__device__ bool GreaterThan(ArbInt *a, ArbInt *b)
{
	for (int ii = 0; ii < DIGITS; ii++)
		if (a->digits[ii] != b->digits[ii])
			return a->digits[ii] > b->digits[ii];

	return false;
}

__device__ bool GreaterThan(ArbHugeInt *a, ArbHugeInt *b)
{
	for (int ii = 0; ii < HUGE_DIGITS; ii++)
		if (a->digits[ii] != b->digits[ii])
			return a->digits[ii] > b->digits[ii];

	return false;
}





/************************************ Mathematical Operations on Arb numbers **************************************************************/

__device__ void AddHugePositiveIntegers(ArbHugeInt *a, ArbHugeInt *b, ArbHugeInt *result)
{
	uint16_t carry = 0;

	result->sign = true;
	bool all_zeros = true;
	for (int ii = 0; ii < HUGE_DIGITS; ii++)
	{
		result->digits[ii] = 0;
		if (all_zeros == true && (a->digits[ii] != 0 || b->digits[ii] != 0))
			all_zeros = false;
	}

	if (all_zeros)
	{
		result->sign = true;
		return;
	}

	for (int ii = HUGE_DIGITS - 1; ii >= 0; ii--)
	{
		int sum = a->digits[ii] + b->digits[ii] + carry;
		if (sum >= MAGNITUDE)
		{
			sum = sum - MAGNITUDE;
			carry = 1;
		}
		else
			carry = 0;

		result->digits[ii] = sum;
	}
}


// Multiply a and b.
__device__ void MultiplyHugePositiveIntegers(ArbInt *a, ArbInt *b, ArbHugeInt *result, ArbHugeInt *each_line, ArbHugeInt *sum_result)
{
	uint16_t carry = 0;
	bool a_on_top = GreaterThan(a, b);
	//	struct ArbHugeInt result;
	for (int kk = 0; kk < HUGE_DIGITS; kk++)
	{
		result->digits[kk] = 0;
		sum_result->digits[kk] = 0;
	}

	int a_size = -1;
	for (int kk = 0; kk < DIGITS; kk++)
	{
		if (a->digits[kk] != 0)
		{
			a_size = kk;
			break;
		}
	}

	int b_size = -1;
	for (int kk = 0; kk < DIGITS; kk++)
	{
		if (b->digits[kk] != 0)
		{
			b_size = kk;
			break;
		}
	}

	if (a_size == -1 || b_size == -1)
	{
		result->sign = true;
		return;
	}


	a_size -= 1;
	b_size -= 1;
	int top_size = (a_on_top ? a_size : b_size);
	int bot_size = (a_on_top ? b_size : a_size);
	for (int ii = DIGITS - 1; ii > bot_size; ii--)
	{
		int idx_move = DIGITS - ii - 1;
		//struct ArbHugeInt thisline;
		for (int kk = 0; kk < HUGE_DIGITS; kk++)
			each_line->digits[kk] = 0;

		each_line->sign = true;

		for (int jj = DIGITS - 1; jj > top_size; jj--)
		{
			int prod = (a_on_top ? (a->digits[jj] * b->digits[ii]) : (a->digits[ii] * b->digits[jj])) + carry;
			if (prod >= MAGNITUDE)
			{
				carry = prod / MAGNITUDE;
				prod = prod - (carry * MAGNITUDE);
			}
			else
			{
				carry = 0;
			}

			int idx = jj - idx_move + DIGITS;
			each_line->digits[idx] = prod;
		}

		if (carry > 0)
		{
			int idx = top_size - idx_move + DIGITS;
			each_line->digits[idx] = carry;
		}

		AddHugePositiveIntegers(sum_result, each_line, result);

		// Copy result back onto sum_result.
		for (int kk = 0; kk < HUGE_DIGITS; kk++)
		{
			sum_result->digits[kk] = result->digits[kk];
		}
	}

	bool a_sign = a->sign;
	bool b_sign = b->sign;
	if (a_sign == false && b_sign == false)
		result->sign = true;
	else if (a_sign == false || b_sign == false)
		result->sign = false;
}



__device__ void AddTwoPositiveArbitrary(ArbDec *a, ArbDec *b, ArbDec *result)
{
	uint16_t carry = 0;
	//struct ArbDec result;
	for (int ii = 0; ii < DIGITS; ii++)
		result->digits[ii] = 0;

	result->sign = true;
	result->decpos = a->decpos > b->decpos ? a->decpos : b->decpos;

	int pos_diff = a->decpos - b->decpos;
	bool a_on_top = GreaterThan(a, b);
	if (pos_diff < 0)
		pos_diff = -pos_diff;

	for (int ii = DMO; ii >= 0; ii--)
	{
		int bot_idx = ii - pos_diff;
		int sum = carry + (a_on_top ? a->digits[ii] : b->digits[ii]) + (bot_idx < 0 ? 0 : (a_on_top ? b->digits[bot_idx] : a->digits[bot_idx]));
		if (sum >= MAGNITUDE)
		{
			carry = 1;
			sum = sum - MAGNITUDE;
		}
		else
			carry = 0;

		result->digits[ii] = sum;
	}

	if (carry > 0)
	{
		// Shift the array right to make room for the carry.
		for (int jj = DMO - 1; jj > 0; jj--)
		{
			result->digits[jj] = result->digits[jj - 1];
		}
		result->digits[0] = carry;
		result->decpos = result->decpos + 1;
	}
}


__device__ void SubtractTwoPositiveArbitrary(ArbDec *a, ArbDec *b, ArbDec *result)
{
	uint16_t take = 0;
	//struct ArbDec result;
	bool all_zeros = true;
	for (int ii = 0; ii < DIGITS; ii++)
	{
		result->digits[ii] = 0;
		if (all_zeros == true && (a->digits[ii] != 0 || b->digits[ii] != 0))
			all_zeros = false;
	}

	if (all_zeros)
	{
		result->sign = true;
		result->decpos = 0;
		return;
	}

	result->sign = true;
	result->decpos = a->decpos > b->decpos ? a->decpos : b->decpos;


	int pos_diff = a->decpos - b->decpos;
	bool a_on_top = GreaterThan(a, b);
	if (a_on_top == false)
		result->sign = false;

	if (pos_diff < 0)
		pos_diff = -pos_diff;

	for (int ii = DMO; ii >= 0; ii--)
	{
		int bot_idx = ii - pos_diff;
		int sum = (a_on_top ? a->digits[ii] : b->digits[ii]) - (bot_idx < 0 ? 0 : (a_on_top ? b->digits[bot_idx] : a->digits[bot_idx])) - take;
		if (sum < 0)
		{
			take = 1;
			sum += MAGNITUDE;
		}
		else
			take = 0;

		result->digits[ii] = sum;
	}

	if (result->digits[0] == 0 && result->decpos > 0)
	{
		// Shift the array left to cinch up the zero.
		for (int jj = 0; jj < DMO - 1; jj++)
		{
			result->digits[jj] = result->digits[jj + 1];
		}
		result->digits[DMO] = 0;
		result->decpos = result->decpos - 1;
	}
}


__device__ void MultiplyArbitrary(ArbDec *a, ArbDec *b, ArbDec *result, ArbInt *a_int, ArbInt *b_int, ArbHugeInt *huge_scratch, ArbHugeInt *each_line, ArbHugeInt *sum_result)
{

	// OPTIMIZE: If a or b is zero, return zero.
	// OPTIMIZE: If a or b is one, return the other one.

	//struct ArbDec result;
	result->sign = true;
	for (int kk = 0; kk < DIGITS; kk++)
		result->digits[kk] = 0;

	bool a_on_top = GreaterThan(a, b);

	//struct ArbInt a_int;
	a_int->sign = a->sign;

	//struct ArbInt b_int;
	b_int->sign = b->sign;

	// Shift everything RIGHT size-number of digits so we have overflow.
	// This is the precision penalty paid when digits approach the max digits.
	int a_zeroes = -1;
	for (int jj = DMO; jj >= 0; jj--)
	{
		if (a->digits[jj] != 0)
		{
			a_zeroes = DMO - jj;
			break;
		}
	}

	int b_zeroes = -1;
	for (int jj = DMO; jj >= 0; jj--)
	{
		if (b->digits[jj] != 0)
		{
			b_zeroes = DMO - jj;
			break;
		}
	}



	// OPTIMIZED: IF EITHER a and b are zero, just return zero.
	if (a_zeroes == -1 || b_zeroes == -1)
	{
		return;
	}


	// Shift everything right to make it useful for integer math.

	for (int jj = DMO; jj >= 0; jj--)
	{
		if (jj >= a_zeroes)
		{
			a_int->digits[jj] = a->digits[jj - a_zeroes];
		}
		else
		{
			a_int->digits[jj] = 0;
		}

		if (jj >= b_zeroes)
		{
			b_int->digits[jj] = b->digits[jj - b_zeroes];
		}
		else
		{
			b_int->digits[jj] = 0;
		}
	}


	if (a_on_top)
		MultiplyHugePositiveIntegers(a_int, b_int, huge_scratch, each_line, sum_result);
	else
		MultiplyHugePositiveIntegers(b_int, a_int, huge_scratch, each_line, sum_result);

	int size = (DMO - a_zeroes) + (DMO - b_zeroes);

	result->sign = huge_scratch->sign;

	for (int jj = 0; jj <= DMO; jj++)
	{
		int idx_huge = HDMO - size + jj - 1;
		if (idx_huge <= HDMO)
			result->digits[jj] = huge_scratch->digits[idx_huge];
		else
			result->digits[jj] = 0;
	}
	result->decpos = a->decpos + b->decpos;


	// Now trim off leading zeros by shifting left.
	int result_non_zero_idx = -1;
	for (int jj = 0; jj < DMO; jj++)
	{
		if (result->digits[jj] != 0)
		{
			result_non_zero_idx = jj;
			break;
		}
	}

	if (result_non_zero_idx == -1)
	{
		result->decpos = 0;
		result->sign = true;
	}
	else
	{
		if (result_non_zero_idx > result->decpos)
			result_non_zero_idx = result->decpos;

		if (result_non_zero_idx > 0)
		{
			int over = DMO - result_non_zero_idx;
			for (int jj = 0; jj < DMO; jj++)
			{
				result->digits[jj] = jj > over ? 0 : result->digits[jj + result_non_zero_idx];
			}
			result->decpos = result->decpos - result_non_zero_idx;

			if (result->decpos < 0)
				result->decpos = 0;
		}
	}

	bool a_sign = a->sign;
	bool b_sign = b->sign;
	if (a_sign == false && b_sign == false)
		result->sign = true;
	else if (a_sign == false || b_sign == false)
		result->sign = false;
}



__device__ void Add(ArbDec *a, ArbDec *b, ArbDec *result)
{
	// -a + b = b - a
	if (a->sign == false && b->sign == true)
	{
		SubtractTwoPositiveArbitrary(b, a, result);
	}
	// a + -b = a - b
	else if (a->sign == true && b->sign == false)
	{
		SubtractTwoPositiveArbitrary(a, b, result);
	}
	// a + b = a + b
	else if (a->sign == true && b->sign == true)
	{
		AddTwoPositiveArbitrary(a, b, result);
	}
	// -a + -b = -(a+b)
	else if (a->sign == false && b->sign == false)
	{
		AddTwoPositiveArbitrary(a, b, result);
		result->sign = false;
		result;
	}
}



__device__ void Subtract(ArbDec *a, ArbDec *b, ArbDec *result)
{
	// -a - b = -(a+b)
	if (a->sign == false && b->sign == true)
	{
		AddTwoPositiveArbitrary(a, b, result);
		result->sign = false;
	}
	// a - -b = a + b
	else if (a->sign == true && b->sign == false)
	{
		AddTwoPositiveArbitrary(a, b, result);
	}
	// a - b = a - b
	else if (a->sign == true && b->sign == true)
	{
		SubtractTwoPositiveArbitrary(a, b, result);
	}
	// -a - -b = -a + b = b - a
	else if (a->sign == false && b->sign == false)
	{
		SubtractTwoPositiveArbitrary(b, a, result);
	}
}


__device__ bool KeepGoing(ArbDec *x, ArbDec *y, ArbDec *x2, ArbDec *y2, ArbDec *sum, ArbInt *a_int, ArbInt *b_int, ArbHugeInt *huge_scratch, ArbHugeInt *each_line, ArbHugeInt *sum_result)
{
	// x ^ 2
	MultiplyArbitrary(x, x, x2, a_int, b_int, huge_scratch, each_line, sum_result);

	// y ^ 2
	MultiplyArbitrary(y, y, y2, a_int, b_int, huge_scratch, each_line, sum_result);

	// sum the squares
	Add(x2, y2, sum);

	// Keep going if the sum is less than four.
	bool result = sum->decpos == 0 ? true : sum->digits[sum->decpos - 1] < 4;
	return result;
}

__device__ void TimesTwo(ArbDec *a, ArbDec *result, ArbDec *two, ArbInt *a_int, ArbInt *b_int, ArbHugeInt *huge_scratch, ArbHugeInt *each_line, ArbHugeInt *sum_result)
{
	MultiplyArbitrary(a, two, result, a_int, b_int, huge_scratch, each_line, sum_result);
}


__global__ void kernel(double** map, int *c, int xsize,
	int y_scale_sign, int y_scale_decpos, uint16_t *y_scale_digits,
	int  y_base_sign, int y_base_decpos, uint16_t *y_base_digits,
	int  x_scale_sign, int x_scale_decpos, uint16_t *x_scale_digits,
	int  x_base_sign, int x_base_decpos, uint16_t *x_base_digits,
	int max)
{

	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = xsize * idx_y + idx_x;


	// The number two!
	ArbDec two = GetZeroDec();
	two.sign = true;
	two.decpos = 1;
	two.digits[0] = 2;


	// Indexes.. where are we?
	ArbDec idx_x_arb = ArbFromDouble((double)idx_x);
	ArbDec idx_y_arb = ArbFromDouble((double)idx_y);


	// Contract the bases and scales.
	ArbDec x_base = Construct(x_base_sign, x_base_decpos, x_base_digits);
	ArbDec x_scale = Construct(x_scale_sign, x_scale_decpos, x_scale_digits);
	ArbDec y_base = Construct(y_base_sign, y_base_decpos, y_base_digits);
	ArbDec y_scale = Construct(y_scale_sign, y_scale_decpos, y_scale_digits);


	// Scratchpad vars for small memory footprint.
	// DONT use GetZeroDec here because i want to do a single loop.
	ArbDec x0;
	ArbDec x_mult;
	ArbDec y0;
	ArbDec y_mult;
	ArbDec x;
	ArbDec y;
	ArbInt a_int;
	ArbInt b_int;
	ArbHugeInt huge_scratch;
	ArbHugeInt each_line;
	ArbHugeInt sum_result;
	ArbDec x2;
	ArbDec y2;
	ArbDec sum;
	ArbDec diff;
	ArbDec temp;
	ArbDec x_times_2;
	ArbDec x_times_2_times_y;


	for (int kk = 0; kk < DIGITS; kk++)
	{
		x0.sign = true;
		y0.sign = true;
		x_mult.sign = true;
		y_mult.sign = true;
		x.sign = true;
		y.sign = true;
		a_int.sign = true;
		b_int.sign = true;
		huge_scratch.sign = true;
		each_line.sign = true;
		sum_result.sign = true;
		x2.sign = true;
		y2.sign = true;
		sum.sign = true;
		diff.sign = true;
		temp.sign = true;
		x_times_2.sign = true;
		x_times_2_times_y.sign = true;


		x0.digits[kk] = 0;
		y0.digits[kk] = 0;
		x_mult.digits[kk] = 0;
		y_mult.digits[kk] = 0;
		x.digits[kk] = 0;
		y.digits[kk] = 0;
		a_int.digits[kk] = 0;
		b_int.digits[kk] = 0;
		huge_scratch.digits[kk] = 0;
		each_line.digits[kk] = 0;
		sum_result.digits[kk] = 0;
		x2.digits[kk] = 0;
		y2.digits[kk] = 0;
		sum.digits[kk] = 0;
		diff.digits[kk] = 0;
		temp.digits[kk] = 0;
		x_times_2.digits[kk] = 0;
		x_times_2_times_y.digits[kk] = 0;
	}

	// Calculate x0 from base and scale
	MultiplyArbitrary(&idx_x_arb, &x_scale, &x_mult, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
	Add(&x_base, &x_mult, &x0);

	// Calculate y0 from base and scale.
	MultiplyArbitrary(&idx_y_arb, &y_scale, &y_mult, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
	Add(&y_base, &y_mult, &y0);


	int ii = 0;
	while (KeepGoing(&x, &y, &x2, &y2, &sum, &a_int, &b_int, &huge_scratch, &each_line, &sum_result) && ii < max)
	{
		// Calculate x
		MultiplyArbitrary(&x, &x, &x2, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
		MultiplyArbitrary(&y, &y, &y2, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
		Subtract(&x2, &y2, &diff);
		Add(&diff, &x0, &temp);

		// Calculate y
		TimesTwo(&x, &x_times_2, &two, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
		MultiplyArbitrary(&x_times_2, &y, &x_times_2_times_y, &a_int, &b_int, &huge_scratch, &each_line, &sum_result);
		Add(&x_times_2_times_y, &y0, &y);

		// Move temp into x.
		for (int kk = 0; kk < DIGITS; kk++)
			x.digits[kk] = temp.digits[kk];


		// Increase the iterator!!
		ii++;
	}

	//printf("%d took %d iterations\n", idx, ii);

	c[idx] = ii;
}

int main()
{
	uint32_t j = 0;
	return 0;
}